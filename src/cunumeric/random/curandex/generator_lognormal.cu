// (c) 2022 NVIDIA all rights reserved
#include "generator.cuh"

template <typename field_t>
struct lognormal_t;

template <>
struct lognormal_t<float> {
  float mean   = 0.0;
  float stddev = 1.0;

  template <typename gen_t>
  __forceinline__ __host__ __device__ float operator()(gen_t& gen)
  {
    return hiprand_log_normal(&gen, mean, stddev);
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateLogNormalEx(
  curandGeneratorEx_t generator, float* outputPtr, size_t n, float mean, float stddev)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  lognormal_t<float> func;
  func.mean   = mean;
  func.stddev = stddev;
  return curandimpl::dispatch_sample<lognormal_t<float>, float>(gen, func, n, outputPtr);
}

template <>
struct lognormal_t<double> {
  double mean   = 0.0;
  double stddev = 1.0;

  template <typename gen_t>
  __forceinline__ __host__ __device__ double operator()(gen_t& gen)
  {
    return hiprand_log_normal_double(&gen, mean, stddev);
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateLogNormalDoubleEx(
  curandGeneratorEx_t generator, double* outputPtr, size_t n, double mean, double stddev)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  lognormal_t<double> func;
  func.mean   = mean;
  func.stddev = stddev;
  return curandimpl::dispatch_sample<lognormal_t<double>, double>(gen, func, n, outputPtr);
}