#include "hip/hip_runtime.h"
// (c) 2022 NVIDIA all rights reserved
#include "generator.cuh"

template <typename field_t>
struct gumbel_t;

template <>
struct gumbel_t<float> {
  float mu, beta;

  // gumble cdf : $ cdf(x) = \exp^{-\exp^{-\frac{x-\mu}{\beta}}} $
  template <typename gen_t>
  __forceinline__ __host__ __device__ float operator()(gen_t& gen)
  {
    float y = hiprand_uniform(&gen);  // y cannot be zero
    if (y == 1.0f) return mu;
    float lny = ::logf(y);
    return mu - beta * ::logf(-lny);
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateGumbelEx(
  curandGeneratorEx_t generator, float* outputPtr, size_t num, float mu, float beta)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  gumbel_t<float> func;
  func.mu   = mu;
  func.beta = beta;
  return curandimpl::dispatch_sample<gumbel_t<float>, float>(gen, func, num, outputPtr);
}

template <>
struct gumbel_t<double> {
  double mu, beta;

  template <typename gen_t>
  __forceinline__ __host__ __device__ double operator()(gen_t& gen)
  {
    double y = hiprand_uniform_double(&gen);  // y cannot be zero
    if (y == 1.0) return mu;
    double lny = ::log(y);
    return mu - beta * ::log(-lny);
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateGumbelDoubleEx(
  curandGeneratorEx_t generator, double* outputPtr, size_t num, double mu, double beta)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  gumbel_t<double> func;
  func.mu   = mu;
  func.beta = beta;
  return curandimpl::dispatch_sample<gumbel_t<double>, double>(gen, func, num, outputPtr);
}