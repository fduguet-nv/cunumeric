#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "generator.cuh"

template <typename field_t>
struct power_t;

template <>
struct power_t<float> {
  float xm, invalpha;

  template <typename gen_t>
  __forceinline__ __host__ __device__ float operator()(gen_t& gen)
  {
    float y = hiprand_uniform(&gen);  // y cannot be 0
    return ::expf(::logf(y) * invalpha);
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGeneratePowerEx(curandGeneratorEx_t generator,
                                                          float* outputPtr,
                                                          size_t num,
                                                          float alpha)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  power_t<float> func;
  func.invalpha = 1.0f / alpha;
  return curandimpl::dispatch_sample<power_t<float>, float>(gen, func, num, outputPtr);
}

template <>
struct power_t<double> {
  double xm, invalpha;

  template <typename gen_t>
  __forceinline__ __host__ __device__ double operator()(gen_t& gen)
  {
    double y = hiprand_uniform_double(&gen);  // y cannot be 0 -- use y as 1-cdf(x)
    return ::exp(::log(y) * invalpha);
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGeneratePowerDoubleEx(curandGeneratorEx_t generator,
                                                                double* outputPtr,
                                                                size_t num,
                                                                double alpha)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  power_t<double> func;
  func.invalpha = 1.0 / alpha;
  return curandimpl::dispatch_sample<power_t<double>, double>(gen, func, num, outputPtr);
}