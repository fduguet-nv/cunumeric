/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "generator.cuh"

template <typename gen_t, curandimpl::execlocation location>
hiprandStatus_t createGeneratorEx(curandGeneratorEx_t* generator,
                                 uint64_t seed,
                                 uint64_t generatorID,
                                 hipStream_t stream = nullptr)
{
  try {
    curandimpl::inner_generator<gen_t, location>* result =
      new curandimpl::inner_generator<gen_t, location>(seed, generatorID, stream);
    *generator = (curandGeneratorEx_t)result;
    return HIPRAND_STATUS_SUCCESS;
  } catch (int errorCode) {
    return (hiprandStatus_t)errorCode;
  }
}

template <curandimpl::execlocation location>
static hiprandStatus_t CURANDAPI inner_curandCreateGeneratorEx(curandGeneratorEx_t* generator,
                                                              hiprandRngType_t rng_type,
                                                              uint64_t seed,
                                                              uint64_t generatorID,
                                                              hipStream_t stream = nullptr)
{
  switch (rng_type) {
    case HIPRAND_RNG_PSEUDO_XORWOW:
      return createGeneratorEx<hiprandStateXORWOW_t, location>(generator, seed, generatorID, stream);
    case HIPRAND_RNG_PSEUDO_PHILOX4_32_10:
      return createGeneratorEx<hiprandStatePhilox4_32_10_t, location>(
        generator, seed, generatorID, stream);
    case HIPRAND_RNG_PSEUDO_MRG32K3A:
      return createGeneratorEx<hiprandStateMRG32k3a_t, location>(
        generator, seed, generatorID, stream);
    default: return HIPRAND_STATUS_TYPE_ERROR;
  }
}

extern "C" hiprandStatus_t CURANDAPI curandCreateGeneratorEx(curandGeneratorEx_t* generator,
                                                            hiprandRngType_t rng_type,
                                                            uint64_t seed,
                                                            uint64_t generatorID,
                                                            uint32_t flags,
                                                            hipStream_t stream)
{
  return inner_curandCreateGeneratorEx<curandimpl::execlocation::DEVICE>(
    generator, rng_type, seed, generatorID, stream);
}

extern "C" hiprandStatus_t CURANDAPI curandCreateGeneratorHostEx(curandGeneratorEx_t* generator,
                                                                hiprandRngType_t rng_type,
                                                                uint64_t seed,
                                                                uint64_t generatorID,
                                                                uint32_t flags)
{
  return inner_curandCreateGeneratorEx<curandimpl::execlocation::HOST>(
    generator, rng_type, seed, generatorID, nullptr);
}

extern "C" hiprandStatus_t CURANDAPI curandDestroyGeneratorEx(curandGeneratorEx_t generator)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  try {
    gen->destroy();
    delete gen;

    return HIPRAND_STATUS_SUCCESS;
  } catch (int errorCode) {
    delete gen;
    return (hiprandStatus_t)errorCode;
  }
}