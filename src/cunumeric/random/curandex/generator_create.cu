// (c) 2022 NVIDIA all rights reserved
#include "generator.cuh"

template <typename gen_t, curandimpl::execlocation location>
hiprandStatus_t createGeneratorEx(curandGeneratorEx_t* generator,
                                 uint64_t seed,
                                 uint64_t generatorID,
                                 hipStream_t stream = nullptr)
{
  try {
    curandimpl::inner_generator<gen_t, location>* result =
      new curandimpl::inner_generator<gen_t, location>(seed, generatorID, stream);
    *generator = (curandGeneratorEx_t)result;
    return HIPRAND_STATUS_SUCCESS;
  } catch (int errorCode) {
    return (hiprandStatus_t)errorCode;
  }
}

template <curandimpl::execlocation location>
static hiprandStatus_t CURANDAPI inner_curandCreateGeneratorEx(curandGeneratorEx_t* generator,
                                                              hiprandRngType_t rng_type,
                                                              uint64_t seed,
                                                              uint64_t generatorID,
                                                              hipStream_t stream = nullptr)
{
  switch (rng_type) {
    case HIPRAND_RNG_PSEUDO_XORWOW:
      return createGeneratorEx<hiprandStateXORWOW_t, location>(generator, seed, generatorID, stream);
    case HIPRAND_RNG_PSEUDO_PHILOX4_32_10:
      return createGeneratorEx<hiprandStatePhilox4_32_10_t, location>(
        generator, seed, generatorID, stream);
    case HIPRAND_RNG_PSEUDO_MRG32K3A:
      return createGeneratorEx<hiprandStateMRG32k3a_t, location>(
        generator, seed, generatorID, stream);
    default: return HIPRAND_STATUS_TYPE_ERROR;
  }
}

extern "C" hiprandStatus_t CURANDAPI curandCreateGeneratorEx(curandGeneratorEx_t* generator,
                                                            hiprandRngType_t rng_type,
                                                            uint64_t seed,
                                                            uint64_t generatorID,
                                                            uint32_t flags,
                                                            hipStream_t stream)
{
  return inner_curandCreateGeneratorEx<curandimpl::execlocation::DEVICE>(
    generator, rng_type, seed, generatorID, stream);
}

extern "C" hiprandStatus_t CURANDAPI curandCreateGeneratorHostEx(curandGeneratorEx_t* generator,
                                                                hiprandRngType_t rng_type,
                                                                uint64_t seed,
                                                                uint64_t generatorID,
                                                                uint32_t flags)
{
  return inner_curandCreateGeneratorEx<curandimpl::execlocation::HOST>(
    generator, rng_type, seed, generatorID, nullptr);
}

extern "C" hiprandStatus_t CURANDAPI curandDestroyGeneratorEx(curandGeneratorEx_t generator)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  try {
    gen->destroy();
    delete gen;

    return HIPRAND_STATUS_SUCCESS;
  } catch (int errorCode) {
    delete gen;
    return (hiprandStatus_t)errorCode;
  }
}