#include "hip/hip_runtime.h"
// (c) 2022 NVIDIA all rights reserved
#include "generator.cuh"

template <typename field_t>
struct logistic_t;

template <>
struct logistic_t<float> {
  float mu, beta;

  template <typename gen_t>
  __forceinline__ __host__ __device__ float operator()(gen_t& gen)
  {
    float y = hiprand_uniform(&gen);  // y cannot be 0
    float t = 1.0f / y - 1.0f;
    if (t == 0) t = 1.0f;
    return mu - beta * ::logf(t);
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateLogisticEx(
  curandGeneratorEx_t generator, float* outputPtr, size_t num, float mu, float beta)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  logistic_t<float> func;
  func.mu   = mu;
  func.beta = beta;
  return curandimpl::dispatch_sample<logistic_t<float>, float>(gen, func, num, outputPtr);
}

template <>
struct logistic_t<double> {
  double mu, beta;

  template <typename gen_t>
  __forceinline__ __host__ __device__ double operator()(gen_t& gen)
  {
    float y = hiprand_uniform_double(&gen);  // y cannot be 0
    float t = 1.0 / y - 1.0;
    if (t == 0) t = 1.0;
    return mu - beta * ::log(t);
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateLogisticDoubleEx(
  curandGeneratorEx_t generator, double* outputPtr, size_t num, double mu, double beta)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  logistic_t<double> func;
  func.mu   = mu;
  func.beta = beta;
  return curandimpl::dispatch_sample<logistic_t<double>, double>(gen, func, num, outputPtr);
}