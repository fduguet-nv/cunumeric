#include "hip/hip_runtime.h"
// (c) 2022 NVIDIA all rights reserved
#include "generator.cuh"

template <typename field_t>
struct rayleigh_t;

template <>
struct rayleigh_t<float> {
  float sigma;

  template <typename gen_t>
  __forceinline__ __host__ __device__ float operator()(gen_t& gen)
  {
    float y = hiprand_uniform(&gen);  // y cannot be 0
    return sigma * ::sqrtf(-2.0f * ::logf(y));
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateRayleighEx(curandGeneratorEx_t generator,
                                                             float* outputPtr,
                                                             size_t num,
                                                             float sigma)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  rayleigh_t<float> func;
  func.sigma = sigma;
  return curandimpl::dispatch_sample<rayleigh_t<float>, float>(gen, func, num, outputPtr);
}

template <>
struct rayleigh_t<double> {
  double sigma;

  template <typename gen_t>
  __forceinline__ __host__ __device__ double operator()(gen_t& gen)
  {
    double y = hiprand_uniform_double(&gen);  // y cannot be 0
    return sigma * ::sqrt(-2.0 * ::log(y));
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateRayleighDoubleEx(curandGeneratorEx_t generator,
                                                                   double* outputPtr,
                                                                   size_t num,
                                                                   double sigma)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  rayleigh_t<double> func;
  func.sigma = sigma;
  return curandimpl::dispatch_sample<rayleigh_t<double>, double>(gen, func, num, outputPtr);
}
