#include "hip/hip_runtime.h"
/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "generator.cuh"

template <typename field_t>
struct weilbull_t;

template <>
struct weilbull_t<float> {
  float lambda, invk;

  template <typename gen_t>
  __forceinline__ __host__ __device__ float operator()(gen_t& gen)
  {
    float y = hiprand_uniform(&gen);  // y cannot be 0
    // log(y) can be zero !
    float lny = ::logf(y);
    if (lny == 0.0f) return 0.0f;
    return lambda * ::expf(::logf(-lny) * invk);
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateWeibullEx(
  curandGeneratorEx_t generator, float* outputPtr, size_t num, float lambda, float k)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  weilbull_t<float> func;
  func.lambda = lambda;
  func.invk   = 1.0f / k;
  return curandimpl::dispatch_sample<weilbull_t<float>, float>(gen, func, num, outputPtr);
}

template <>
struct weilbull_t<double> {
  double lambda, invk;

  template <typename gen_t>
  __forceinline__ __host__ __device__ double operator()(gen_t& gen)
  {
    double y = hiprand_uniform_double(&gen);  // y cannot be 0
    // log(y) can be zero !
    float lny = ::log(y);
    if (lny == 0.0f) return 0.0f;
    return lambda * ::exp(::log(-lny) * invk);
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateWeibullDoubleEx(
  curandGeneratorEx_t generator, double* outputPtr, size_t num, double lambda, double k)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  weilbull_t<double> func;
  func.lambda = lambda;
  func.invk   = 1.0 / k;
  return curandimpl::dispatch_sample<weilbull_t<double>, double>(gen, func, num, outputPtr);
}
