#include "hip/hip_runtime.h"
// (c) 2022 NVIDIA all rights reserved
#include "generator.cuh"

template <typename field_t>
struct exponential_t;

template <>
struct exponential_t<float> {
  float scale = 1.0f;

  template <typename gen_t>
  __forceinline__ __host__ __device__ float operator()(gen_t& gen)
  {
    float uni = hiprand_uniform(&gen);
    return -::logf(uni) * scale;
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateExponentialEx(curandGeneratorEx_t generator,
                                                                float* outputPtr,
                                                                size_t n,
                                                                float scale)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  exponential_t<float> func;
  func.scale = scale;
  return curandimpl::dispatch_sample<exponential_t<float>, float>(gen, func, n, outputPtr);
}

template <>
struct exponential_t<double> {
  double scale = 1.0f;

  template <typename gen_t>
  __forceinline__ __host__ __device__ double operator()(gen_t& gen)
  {
    double uni = hiprand_uniform_double(&gen);
    return -::logf(uni) * scale;
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateExponentialDoubleEx(curandGeneratorEx_t generator,
                                                                      double* outputPtr,
                                                                      size_t n,
                                                                      double scale)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  exponential_t<double> func;
  func.scale = scale;
  return curandimpl::dispatch_sample<exponential_t<double>, double>(gen, func, n, outputPtr);
}
