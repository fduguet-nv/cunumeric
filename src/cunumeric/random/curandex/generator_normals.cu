// (c) 2022 NVIDIA all rights reserved
#include "generator.cuh"

template <typename field_t>
struct normal_t;

template <>
struct normal_t<float> {
  float mean   = 0.0;
  float stddev = 1.0;

  template <typename gen_t>
  __forceinline__ __host__ __device__ float operator()(gen_t& gen)
  {
    return stddev * hiprand_normal(&gen) + mean;
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateNormalEx(
  curandGeneratorEx_t generator, float* outputPtr, size_t n, float mean, float stddev)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  normal_t<float> func;
  func.mean   = mean;
  func.stddev = stddev;
  return curandimpl::dispatch_sample<normal_t<float>, float>(gen, func, n, outputPtr);
}

template <>
struct normal_t<double> {
  double mean   = 0.0;
  double stddev = 1.0;

  template <typename gen_t>
  __forceinline__ __host__ __device__ double operator()(gen_t& gen)
  {
    return stddev * hiprand_normal_double(&gen) + mean;
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateNormalDoubleEx(
  curandGeneratorEx_t generator, double* outputPtr, size_t n, double mean, double stddev)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  normal_t<double> func;
  func.mean   = mean;
  func.stddev = stddev;
  return curandimpl::dispatch_sample<normal_t<double>, double>(gen, func, n, outputPtr);
}