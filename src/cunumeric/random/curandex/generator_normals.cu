/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "generator.cuh"

template <typename field_t>
struct normal_t;

template <>
struct normal_t<float> {
  float mean   = 0.0;
  float stddev = 1.0;

  template <typename gen_t>
  __forceinline__ __host__ __device__ float operator()(gen_t& gen)
  {
    return stddev * hiprand_normal(&gen) + mean;
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateNormalEx(
  curandGeneratorEx_t generator, float* outputPtr, size_t n, float mean, float stddev)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  normal_t<float> func;
  func.mean   = mean;
  func.stddev = stddev;
  return curandimpl::dispatch_sample<normal_t<float>, float>(gen, func, n, outputPtr);
}

template <>
struct normal_t<double> {
  double mean   = 0.0;
  double stddev = 1.0;

  template <typename gen_t>
  __forceinline__ __host__ __device__ double operator()(gen_t& gen)
  {
    return stddev * hiprand_normal_double(&gen) + mean;
  }
};

extern "C" hiprandStatus_t CURANDAPI curandGenerateNormalDoubleEx(
  curandGeneratorEx_t generator, double* outputPtr, size_t n, double mean, double stddev)
{
  curandimpl::basegenerator* gen = (curandimpl::basegenerator*)generator;
  normal_t<double> func;
  func.mean   = mean;
  func.stddev = stddev;
  return curandimpl::dispatch_sample<normal_t<double>, double>(gen, func, n, outputPtr);
}