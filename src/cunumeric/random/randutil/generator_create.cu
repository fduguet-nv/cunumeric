/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "generator.cuh"
#include "generator_create.inl"

extern "C" hiprandStatus_t CURANDAPI randutilCreateGenerator(randutilGenerator_t* generator,
                                                            hiprandRngType_t rng_type,
                                                            uint64_t seed,
                                                            uint64_t generatorID,
                                                            uint32_t flags,
                                                            hipStream_t stream)
{
  return inner_randutilCreateGenerator<randutilimpl::execlocation::DEVICE>(
    generator, rng_type, seed, generatorID, stream);
}
