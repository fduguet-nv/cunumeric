/* Copyright 2021-2022 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "generator.cuh"
#include "generator_create.inl"

extern "C" hiprandStatus_t CURANDAPI randutilCreateGenerator(randutilGenerator_t* generator,
                                                            hiprandRngType_t rng_type,
                                                            uint64_t seed,
                                                            uint64_t generatorID,
                                                            uint32_t flags,
                                                            hipStream_t stream)
{
  return inner_randutilCreateGenerator<randutilimpl::execlocation::DEVICE>(
    generator, rng_type, seed, generatorID, stream);
}

namespace randutilimpl {

// partially specialize dispatcher to enable DEVICE implementation generation
template <typename func_t, typename out_t>
struct dispatcher<randutilimpl::execlocation::DEVICE, func_t, out_t> {
  static hiprandStatus_t run(randutilimpl::basegenerator* gen, func_t func, size_t N, out_t* out)
  {
    return inner_dispatch_sample<randutilimpl::execlocation::DEVICE, func_t, out_t>(
      gen, func, N, out);
  }
};

}  // namespace randutilimpl

// explicit instantiations of distributions
#include "generator_integers.inl"
template struct randutilimpl::
  dispatcher<randutilimpl::execlocation::DEVICE, integers<int32_t>, int32_t>;
template struct randutilimpl::
  dispatcher<randutilimpl::execlocation::DEVICE, integers<int64_t>, int64_t>;
